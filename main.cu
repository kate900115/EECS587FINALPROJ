#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cmath>
#include <stdlib.h> 
#include <hip/hip_runtime.h>
  

#define size 500
#define TreeSize 2000
#define MAX 10000000
#define MIN -10000000
#define theta 0.5
#define G 6.67
#define T 0.01


using namespace std;

__device__ double max_x = MIN;
__device__ double min_x = MAX;
__device__ double max_y = MIN;
__device__ double min_y = MAX;

class body
{
	public:
		// the coordinate of the mass center
		double mass_center_x;
		double mass_center_y;
		// the sum of the mass of the sub node
		double mass_sum;
		// array index that store the data
		double array_num;
		// the square space 
		double NW_x;
		double NW_y;
		double SE_x;
		double SE_y;
		// the force
		double Fx;
		double Fy;
		//
		int tree_idx;
		
		//constructor
		//mass initialized as -100 indicate the result of the center of mass is not computed
		//array_num initialized as -1 indicate it don't contain any body
		//array_num = -2 indicate it contains an empty body
		//array_num = -3 indicate it contains a body which has children
		//array_num >=0 indicate it contains a leaf node
		
		__device__ body()
		{
			// the coordinate of the mass center
			mass_center_x = -100; 
			mass_center_y = -100;
			// the sum of the mass of the sub node
			mass_sum = -100; 
			// array index that store the data
			array_num = -1;
			// tree index
			tree_idx = -1;
			// the square space 
			NW_x = min_x;
			NW_y = max_y;
			SE_x = max_x;
			SE_y = min_y;
			// the force 
			Fx = 0;
			Fy = 0;
		}

};


struct Lock
{
	int *mutex;
	//construnctor
	Lock()
	{
		int state = 0;
		hipMalloc((void**)&mutex, sizeof(int));
		hipMemcpy(mutex, &state, sizeof(int), hipMemcpyHostToDevice);
	}

	~Lock()
	{
		hipFree(mutex);
	}

	__device__ void lock()
	{
		//0xFFFFFFFF is just a very large number. 
		//The point is no block index can be this big (currently).
		while(atomicCAS(mutex, 0, 1)!= 0);    
	}
	
	__device__ void unlock()
	{
		atomicExch(mutex, 0);
	}
};


__global__ 
void FindEdge(Lock lock1, double* x, double* y)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	
	lock1.lock();
	max_x = x[i]>max_x?x[i]:max_x;
	min_x = x[i]<min_x?x[i]:min_x;
	max_y = y[i]>max_y?y[i]:max_y;
	min_x = y[i]<min_y?y[i]:min_y;
	lock1.unlock();
	
}
/*
__global__ 
void ConstructQuadtree(double* x, double* y, double* m, int* idx)
{
	

	int i = blockIdx.x*blockDim.x + threadIdx.x;

	double x_start = min_x;
	double x_end = max_x;
	double y_start = min_y;
	double y_end = max_y;
	int j=0;
	int test_int=0;
	while (quadtree[j].array_num!=-2)
	{
		cout<<test_int++<<endl;
		cout<<"i="<<i<<", j="<<j<<" ,"<<quadtree[j].array_num<<endl;
		// if there is a hole exist
		if (quadtree[j].array_num == -3)
		{
			if ((x[i]<(x_start+x_end)/2)&&(y[i]<(y_start+y_end)/2))
			{
				j=4*j+1;
				x_end = (x_start+x_end)/2;
				y_end = (y_start+y_end)/2;
			}
			else if ((x[i]<(x_start+x_end)/2)&&(y[i]>=(y_start+y_end)/2))
			{
				j=4*j+2;
				x_end = (x_start+x_end)/2;
				y_start = (y_start+y_end)/2;
			}
			else if ((x[i]>=(x_start+x_end)/2)&&(y[i]<(y_start+y_end)/2))
			{
				j=4*j+3;
				x_start = (x_start+x_end)/2;
				y_end = (y_start+y_end)/2;
			}
			else if ((x[i]>=(x_start+x_end)/2)&&(y[i]>=(y_start+y_end)/2))
			{
				j=4*j+4;
				x_start = (x_start+x_end)/2;
				y_start = (y_start+y_end)/2;
			}
		}
		// if there is a node exist
		else if (quadtree[j].array_num>-1)
		{
			int temp = quadtree[j].array_num;
			quadtree[j].array_num = -3;
			quadtree[j].mass_sum = -100;
			// insert current node to next level
			if ((x[temp]<(x_start+x_end)/2)&&(y[temp]<(y_start+y_end)/2))
			{
				// create the first node
				quadtree[4*j+1].array_num = temp;
				quadtree[4*j+1].mass_sum = m[temp];
				quadtree[4*j+1].mass_center_x = x[temp]; 
				quadtree[4*j+1].mass_center_y = y[temp];
				idx[temp] = 4*j+1;
				quadtree[4*j+1].NW_x = x_start;
				quadtree[4*j+1].NW_y = y_start;
				quadtree[4*j+1].SE_x = (x_start+x_end)/2;
				quadtree[4*j+1].SE_y = (y_start+y_end)/2;
				// create the rest 3 nodes
				quadtree[4*j+2].array_num = -2;
				quadtree[4*j+2].NW_x = x_start;
				quadtree[4*j+2].NW_y = y_start;
				quadtree[4*j+2].SE_x = (x_start+x_end)/2;
				quadtree[4*j+2].SE_y = (y_start+y_end)/2;
			
				quadtree[4*j+3].array_num = -2;
				quadtree[4*j+3].NW_x = x_start;
				quadtree[4*j+3].NW_y = y_start;
				quadtree[4*j+3].SE_x = (x_start+x_end)/2;
				quadtree[4*j+3].SE_y = (y_start+y_end)/2;

				quadtree[4*j+4].array_num = -2;
				quadtree[4*j+4].NW_x = x_start;
				quadtree[4*j+4].NW_y = y_start;
				quadtree[4*j+4].SE_x = (x_start+x_end)/2;
				quadtree[4*j+4].SE_y = (y_start+y_end)/2;
			}
			else if ((x[temp]<(x_start+x_end)/2)&&(y[temp]>=(y_start+y_end)/2))
			{
				// create the first node
				quadtree[4*j+1].array_num = -2;
				quadtree[4*j+1].NW_x = x_start;
				quadtree[4*j+1].NW_y = (y_start+y_end)/2;
				quadtree[4*j+1].SE_x = (x_start+x_end)/2;
				quadtree[4*j+1].SE_y = y_end;
				// create the rest 3 nodes
				quadtree[4*j+2].array_num = temp;
				quadtree[4*j+2].mass_sum = m[temp];
				quadtree[4*j+2].mass_center_x = x[temp]; 
				quadtree[4*j+2].mass_center_y = y[temp];
				idx[temp] = 4*j+2;
				quadtree[4*j+2].NW_x = x_start;
				quadtree[4*j+2].NW_y = (y_start+y_end)/2;
				quadtree[4*j+2].SE_x = (x_start+x_end)/2;
				quadtree[4*j+2].SE_y = y_end;
				
				quadtree[4*j+3].array_num = -2;
				quadtree[4*j+3].NW_x = x_start;
				quadtree[4*j+3].NW_y = (y_start+y_end)/2;
				quadtree[4*j+3].SE_x = (x_start+x_end)/2;
				quadtree[4*j+3].SE_y = y_end;
	
				quadtree[4*j+4].array_num = -2;
				quadtree[4*j+4].NW_x = x_start;
				quadtree[4*j+4].NW_y = (y_start+y_end)/2;
				quadtree[4*j+4].SE_x = (x_start+x_end)/2;
				quadtree[4*j+4].SE_y = y_end;
			}
			else if ((x[temp]>=(x_start+x_end)/2)&&(y[temp]<(y_start+y_end)/2))
			{
				// create the first node
				quadtree[4*j+1].array_num = -2;
				quadtree[4*j+1].NW_x = (x_start+x_end)/2;
				quadtree[4*j+1].NW_y = y_start;
				quadtree[4*j+1].SE_x = x_end;
				quadtree[4*j+1].SE_y = (y_start+y_end)/2;
				// create the rest 3 nodes
				quadtree[4*j+2].array_num = -2;
				quadtree[4*j+2].NW_x = (x_start+x_end)/2;
				quadtree[4*j+2].NW_y = y_start;
				quadtree[4*j+2].SE_x = x_end;
				quadtree[4*j+2].SE_y = (y_start+y_end)/2;

				quadtree[4*j+3].array_num = temp;
				quadtree[4*j+3].mass_sum = m[temp];
				quadtree[4*j+3].mass_center_x = x[temp]; 
				quadtree[4*j+3].mass_center_y = y[temp];
				idx[temp] = 4*j+3;
				quadtree[4*j+3].NW_x = (x_start+x_end)/2;
				quadtree[4*j+3].NW_y = y_start;
				quadtree[4*j+3].SE_x = x_end;
				quadtree[4*j+3].SE_y = (y_start+y_end)/2;
	
				quadtree[4*j+4].array_num = -2;
				quadtree[4*j+4].NW_x = (x_start+x_end)/2;
				quadtree[4*j+4].NW_y = y_start;
				quadtree[4*j+4].SE_x = x_end;
				quadtree[4*j+4].SE_y = (y_start+y_end)/2;
			}
			else if ((x[temp]>=(x_start+x_end)/2)&&(y[temp]>=(y_start+y_end)/2))
			{
				// create the first node
				quadtree[4*j+1].array_num = -2;
				quadtree[4*j+1].NW_x = (x_start+x_end)/2;
				quadtree[4*j+1].NW_y = (y_start+y_end)/2;
				quadtree[4*j+1].SE_x = x_end;
				quadtree[4*j+1].SE_y = y_end;
				// create the rest 3 nodes
				quadtree[4*j+2].array_num = -2;
				quadtree[4*j+2].NW_x = (x_start+x_end)/2;
				quadtree[4*j+2].NW_y = (y_start+y_end)/2;
				quadtree[4*j+2].SE_x = x_end;
				quadtree[4*j+2].SE_y = y_end;
				
				quadtree[4*j+3].array_num = -2;
				quadtree[4*j+3].NW_x = (x_start+x_end)/2;
				quadtree[4*j+3].NW_y = (y_start+y_end)/2;
				quadtree[4*j+3].SE_x = x_end;
				quadtree[4*j+3].SE_y = y_end;
	
				quadtree[4*j+4].array_num = temp;
				quadtree[4*j+4].mass_sum = m[temp];
				quadtree[4*j+4].mass_center_x = x[temp]; 
				quadtree[4*j+4].mass_center_y = y[temp];
				idx[temp] = 4*j+4;
				quadtree[4*j+4].NW_x = (x_start+x_end)/2;
				quadtree[4*j+4].NW_y = (y_start+y_end)/2;
				quadtree[4*j+4].SE_x = x_end;
				quadtree[4*j+4].SE_y = y_end;
			}
			
			if ((x[i]<(x_start+x_end)/2)&&(y[i]<(y_start+y_end)/2))
			{
				j=4*j+1;
				x_end = (x_start+x_end)/2;
				y_end = (y_start+y_end)/2;
			}
			else if ((x[i]<(x_start+x_end)/2)&&(y[i]>=(y_start+y_end)/2))
			{
				j=4*j+2;
				x_end = (x_start+x_end)/2;
				y_start = (y_start+y_end)/2;
			}
			else if ((x[i]>=(x_start+x_end)/2)&&(y[i]<(y_start+y_end)/2))
			{
				j=4*j+3;
				x_start = (x_start+x_end)/2;
				y_end = (y_start+y_end)/2;
			}
			else if ((x[i]>=(x_start+x_end)/2)&&(y[i]>=(y_start+y_end)/2))
			{
				j=4*j+4;
				x_start = (x_start+x_end)/2;
				y_start = (y_start+y_end)/2;
			}
			
		}
		if (quadtree[j].array_num == -2)
		{
			quadtree[j].array_num = i;
			quadtree[j].mass_sum = m[i];
			quadtree[j].mass_center_x = x[i];
			quadtree[j].mass_center_y = y[i];
			idx[i]=j;
		}
	}
}


__global__ 
void UpdateMass()
{
	//update the mass_sum of from the leaf to the top
	for (int i=TreeSize-1; i>-1; i--)
	{
		// if this is a internal node
		// we need to update the mass center
		if ((quadtree[i].array_num==-3)&&(quadtree[i].mass_sum<0))
		{
			//first we need to figure out whether the node is ready to compute 
			//the mass center	
			bool IsReady = true;
			double temp_mass_sum = 0;
			double temp_mass_center_x = 0;
			double temp_mass_center_y = 0;
			for (int j=1; j<5; j++)
			{
				if ((quadtree[4*i+j].array_num>-1)||(quadtree[4*i+j].array_num==-3))
				{
					if (quadtree[4*i+j].mass_sum==-100)
					{
						IsReady = false;
					}
				}
			}
			if (IsReady)
			{
				for (int j=1; j<5; j++)
				{
					if (quadtree[4*i+j].mass_sum>0)
					{
						temp_mass_sum = temp_mass_sum+quadtree[4*i+j].mass_sum;
						temp_mass_center_x = temp_mass_center_x + quadtree[4*i+j].mass_sum * quadtree[4*i+j].mass_center_x;
						temp_mass_center_y = temp_mass_center_y + quadtree[4*i+j].mass_sum * quadtree[4*i+j].mass_center_y;
					}
				}
				temp_mass_center_x = temp_mass_center_x / temp_mass_sum;
				temp_mass_center_y = temp_mass_center_y / temp_mass_sum;

				quadtree[i].mass_sum = temp_mass_sum;
				quadtree[i].mass_center_x = temp_mass_center_x;
				quadtree[i].mass_center_y = temp_mass_center_y;
			}
		}
	}
}




__global__ 
void ComputeForce(double* x, double* y, double* m, int* idx, double* fx, double* fy)
{
	// compute forces acting on each body
	int i=blockIdx.x * blockDim.x + threadIdx.x;

	//int TreeIdx = idx[i];
	// traverse from the root of the quadtree
	// GPU cannot use recursive
	// so we create a stack
	body stack[TreeSize];
	int tail=0;
	stack[0]=quadtree[0];
	tail++;

	while (tail!=0)
	{
		// array_num=-2 meaning that the node is a hole and have no child node
		// delete the node
		if (stack[tail-1].array_num==-2)
		{
			tail--;
		}
		// array_num=-3 meaning that the node have child nodes
		// put all the child node into stack
		else if (stack[tail-1].array_num==-3)
		{
			double s = (stack[tail-1].SE_x - stack[tail-1].NW_x)>(stack[tail-1].SE_y - stack[tail-1].NW_y)? 
		    	           (stack[tail-1].SE_x - stack[tail-1].NW_x):(stack[tail-1].SE_y - stack[tail-1].NW_y);
		
			// distance between the current node and the mass center of the node
			double d = sqrt((x[i]-stack[tail-1].mass_center_x)*(x[i]-stack[tail-1].mass_center_x)
		 	           +(y[i]-stack[tail-1].mass_center_y)*(y[i]-stack[tail-1].mass_center_y));
	
			if (s/d<theta)
			{
				int RealIdx = idx[i];

				//compute the force
				double Forth_x = G * m[i]* stack[tail-1].mass_sum * (x[i]-stack[tail-1].mass_center_x) / (d*d*d); 	
				double Forth_y = G * m[i]* stack[tail-1].mass_sum * (y[i]-stack[tail-1].mass_center_y) / (d*d*d);
				
				quadtree[RealIdx].Fx = quadtree[RealIdx].Fx + Forth_x;
				quadtree[RealIdx].Fy = quadtree[RealIdx].Fy + Forth_y;
				fx[i] = fx[i] + Forth_x;
				fy[i] = fy[i] + Forth_y;
				tail--;
			}
			else
			{
				int new_index = stack[tail-1].tree_idx;
				stack[tail-1]=quadtree[4*new_index+1];
				stack[tail]=quadtree[4*new_index+2];
				stack[tail+1]=quadtree[4*new_index+3];
				stack[tail+2]=quadtree[4*new_index+4];
				tail=tail+3;
			}
		} 
		else if (stack[tail-1].array_num>-1)
		{
			// compute the force directly
			double d = sqrt((x[i]-stack[tail-1].mass_center_x)*(x[i]-stack[tail-1].mass_center_x)
		 	           +(y[i]-stack[tail-1].mass_center_y)*(y[i]-stack[tail-1].mass_center_y));

			int RealIdx = idx[i];
				
			if (i!=stack[tail-1].array_num)
			{
				double Forth_x = G * m[i]* stack[tail-1].mass_sum * (x[i]-stack[tail-1].mass_center_x) / (d*d*d); 	
				double Forth_y = G * m[i]* stack[tail-1].mass_sum * (y[i]-stack[tail-1].mass_center_y) / (d*d*d);

				quadtree[RealIdx].Fx = quadtree[RealIdx].Fx + Forth_x;
				quadtree[RealIdx].Fy = quadtree[RealIdx].Fy + Forth_y;
				fx[i] = fx[i] + Forth_x;
				fy[i] = fy[i] + Forth_y;
			}

			tail--;
		}
	}
}


__global__ 
void UpdateSpeed(double* vx, double* vy, double* x, double* y, double* m, double* fx, double* fy)
{
	// update body position and velocities
	int i=blockIdx.x * blockDim.x + threadIdx.x;
	
	double temp_vx = vx[i] + m[i]/fx[i] * T;
	double temp_vy = vy[i] + m[i]/fy[i] * T;
	double temp_x = x[i] + vx[i]*T + 0.5* m[i]/fx[i]*T*T;
	double temp_y = y[i] + vy[i]*T + 0.5* m[i]/fy[i]*T*T;
	vx[i]= temp_vx;
	vy[i]= temp_vy;
	x[i] = temp_x;
	y[i] = temp_y;
}*/


int main(int argc, char** argv)
{
	// the max coordinate and the minimum coordinate of 
	// the 2D-space
	double max_x=MIN;
	double min_x=MAX;
	double max_y=MIN;
	double min_y=MAX;

	Lock UpdataMaxMin;
	string FileName = string(argv[1]);
	ifstream InFile;
	InFile.open(FileName.c_str());
	
	double *x, *y, *m, *vx, *vy, *fx, *fy;
	int *idx;
	body* quadtree;
	x = (double*)malloc(size*sizeof(double));
	y = (double*)malloc(size*sizeof(double));
	m = (double*)malloc(size*sizeof(double));
	vx = (double*)malloc(size*sizeof(double));
	vy = (double*)malloc(size*sizeof(double));
	fx = (double*)malloc(size*sizeof(double));
	fy = (double*)malloc(size*sizeof(double));
	idx = (int*)malloc(size*sizeof(int));
	quadtree = (body*)malloc(TreeSize*sizeof(body));

	double *d_x, *d_y, *d_m, *d_vx, *d_vy, *d_fx, *d_fy;
	int *d_idx;
	body* d_quadtree;
	hipMalloc(&d_x, size*sizeof(double));
	hipMalloc(&d_y, size*sizeof(double));
	hipMalloc(&d_m, size*sizeof(double));
	hipMalloc(&d_vx, size*sizeof(double));
	hipMalloc(&d_vy, size*sizeof(double));
	hipMalloc(&d_fx, size*sizeof(double));
	hipMalloc(&d_fy, size*sizeof(double));
	hipMalloc(&d_idx, size*sizeof(int));
	hipMalloc(&d_quadtree, TreeSize*sizeof(body));

	

	
	string line;
	int index = 0;
	//read the input file
	//find the max_x, max_y, min_x, min_y
	while (getline(InFile, line))
	{
		stringstream sstr;
		string word;
		sstr << line;
		sstr >> word;
		x[index] = atof(word.c_str());
		sstr >> word;
		y[index] = atof(word.c_str());
		sstr >> word;
		m[index] = atof(word.c_str());
		sstr >> word;
		vx[index] = atof(word.c_str());
		sstr >> word;
		vy[index] = atof(word.c_str());
		fx[index] = 0;
		fy[index] = 0;
		index++;
	}

	InFile.close();	

	hipMemcpy(d_x, x, size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_m, m, size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_vx, vx, size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_vy, vy, size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_fx, fx, size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_fy, fy, size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_idx, idx, size*sizeof(int), hipMemcpyHostToDevice);


	// start to record time
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop);
	hipEventRecord (start, 0);

	Lock UpdateMaxMin;

	FindEdge<<<size/256, 256>>>(UpdateMaxMin,d_x, d_y);
/*
	for (int i=0; i<TreeSize; i++)
	{
		quadtree[i].tree_idx = i;
	}

	//insert each body to the tree structure
	quadtree[0].array_num = 0;
	quadtree[0].mass_sum = d_m[0];
	quadtree[0].NW_x = min_x;
	quadtree[0].NW_y = min_y;
	quadtree[0].SE_x = max_x;
	quadtree[0].SE_y = max_y;
	d_idx[0]=0;

	ConstructQuadtree<<<size/256, 256>>>(d_x, d_y, d_m, d_idx);
	UpdateMass<<<size/256, 256>>>();
	ComputeForce()<<<size/256, 256>>>(d_x, d_y, d_m, d_idx, d_fx, d_fy);
	UpdateSpeed()<<<size/256, 256>>>(d_vx, d_vy, d_x, d_y, d_m, d_fx, d_fy);
*/
	// get the relapsed time

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cout<<"Elapsed time = "<<elapsedTime<<endl;

	hipMemcpy(x, d_x, size*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(y, d_y, size*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(vx, d_vx, size*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(vy, d_vy, size*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(fx, d_fx, size*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(fy, d_fy, size*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(idx, d_idx, size*sizeof(int), hipMemcpyDeviceToHost);	

	return 0;

}
